#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>
#include <cfloat>

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );

using namespace optix;

//
// Bounding box program for programmable convex hull primitive
//
rtDeclareVariable(float3, chull_bbmin, , );
rtDeclareVariable(float3, chull_bbmax, , );
RT_PROGRAM void chull_bounds (int primIdx, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->m_min = chull_bbmin;
  aabb->m_max = chull_bbmax;
}

//
// Intersection program for programmable convex hull primitive
//
rtBuffer<float4> planes;
RT_PROGRAM void chull_intersect(int primIdx)
{
  int n = planes.size();
  float t0 = -FLT_MAX;
  float t1 = FLT_MAX;
  float3 t0_normal = make_float3(0);
  float3 t1_normal = make_float3(0);
  for(int i = 0; i < n && t0 < t1; ++i ) {
    float4 plane = planes[i];
    float3 n = make_float3(plane);
    float  d = plane.w;

    float denom = dot(n, ray.direction);
    float t = -(d + dot(n, ray.origin))/denom;
    if( denom < 0){
      // enter
      if(t > t0){
        t0 = t;
        t0_normal = n;
      }
    } else {
      //exit
      if(t < t1){
        t1 = t;
        t1_normal = n;
      }
    }
  }

  if(t0 > t1)
    return;

  if(rtPotentialIntersection( t0 )){
    shading_normal = geometric_normal = t0_normal;
    rtReportIntersection(0);
  } else if(rtPotentialIntersection( t1 )){
    shading_normal = geometric_normal = t1_normal;
    rtReportIntersection(0);
  }
}
